#include "hip/hip_runtime.h"
/*
 * bvhgpu.cu
 *
 *  Created on: 07 Feb 2011
 *      Author: showroom
 */


/*
 * INCLUDES
 */

#include <cutil_inline.h>

#include "bvhgpu.h"


/*
 * FUNCTIONS
 */

/*
int cu_BVHgpu_primitiveIntersection(BVHgpu_ray *ray, unsigned char *primitive)
{
	BVHgpu_primitiveHeader *header = (BVHgpu_primitiveHeader*)primitive;
	unsigned char *data = primitive + sizeof(BVHgpu_primitiveHeader);

	if (header->type == BVHPRIMITIVE_TYPE_TRIANGLE)
	{
		// Variables
		static float t;
		static float pi[3];
		static BVHPrimitive_triangle *triangle;

		// Get triangle data
		triangle = (BVHPrimitive_triangle*)data;

		// Calculate t
		t = ((triangle->v1[0] - ray->pos[0]) * triangle->normal[0] +
			 (triangle->v1[1] - ray->pos[1]) * triangle->normal[1] +
			 (triangle->v1[2] - ray->pos[2]) * triangle->normal[2]) /
			(ray->dir[0] * triangle->normal[0] +
			 ray->dir[1] * triangle->normal[1] +
			 ray->dir[2] * triangle->normal[2]);

		// Check direction
		if (t < 0.0f)
			return 0;

		if (ray->distance != -1.0f && t > ray->distance)
			return 0;

		// Check intersection
		pi[0] = ray->pos[0] + ray->dir[0] * t;
		pi[1] = ray->pos[1] + ray->dir[1] * t;
		pi[2] = ray->pos[2] + ray->dir[2] * t;

		if (((pi[0] - triangle->v1[0]) * triangle->n12[0] +
			 (pi[1] - triangle->v1[1]) * triangle->n12[1] +
			 (pi[2] - triangle->v1[2]) * triangle->n12[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v2[0]) * triangle->n23[0] +
			 (pi[1] - triangle->v2[1]) * triangle->n23[1] +
			 (pi[2] - triangle->v2[2]) * triangle->n23[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v3[0]) * triangle->n31[0] +
			 (pi[1] - triangle->v3[1]) * triangle->n31[1] +
			 (pi[2] - triangle->v3[2]) * triangle->n31[2]) < 0.0f)
			return 0;

		ray->distance = t;
		return 1;
	}
	else
	{
		return 0;
	}
}*/

__global__ void cu_BVHgpu_primitiveIntersection(unsigned int gpu_nNodes, BVHgpu_node *gpu_nodes, BVHgpu_ray *rays, unsigned char *primitives)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (rays[idx].nodeIndex >= gpu_nNodes)
		return;

	// Variables
	float t;
	float pi[3];
	BVHPrimitive_triangle *triangle;

	BVHgpu_primitiveHeader *header = (BVHgpu_primitiveHeader*)(primitives + gpu_nodes[rays[idx].nodeIndex].primitiveLocation);
	unsigned char *data = (unsigned char*)header + sizeof(BVHgpu_primitiveHeader);

	if (header->type == BVHPRIMITIVE_TYPE_TRIANGLE)
	{
		// Get triangle data
		triangle = (BVHPrimitive_triangle*)data;

		// Calculate t
		t = ((triangle->v1[0] - rays[idx].pos[0]) * triangle->normal[0] +
			 (triangle->v1[1] - rays[idx].pos[1]) * triangle->normal[1] +
			 (triangle->v1[2] - rays[idx].pos[2]) * triangle->normal[2]) /
			(rays[idx].dir[0] * triangle->normal[0] +
			 rays[idx].dir[1] * triangle->normal[1] +
			 rays[idx].dir[2] * triangle->normal[2]);

		// Check direction
		if (t < 0.0f)
		{
			rays[idx].nodeIndex++;
			return;
		}

		if (rays[idx].distance != -1.0f && t > rays[idx].distance)
		{
			rays[idx].nodeIndex++;
			return;
		}

		// Check intersection
		pi[0] = rays[idx].pos[0] + rays[idx].dir[0] * t;
		pi[1] = rays[idx].pos[1] + rays[idx].dir[1] * t;
		pi[2] = rays[idx].pos[2] + rays[idx].dir[2] * t;

		if (((pi[0] - triangle->v1[0]) * triangle->n12[0] +
			 (pi[1] - triangle->v1[1]) * triangle->n12[1] +
			 (pi[2] - triangle->v1[2]) * triangle->n12[2]) < 0.0f)
		{
			rays[idx].nodeIndex++;
			return;
		}

		if (((pi[0] - triangle->v2[0]) * triangle->n23[0] +
			 (pi[1] - triangle->v2[1]) * triangle->n23[1] +
			 (pi[2] - triangle->v2[2]) * triangle->n23[2]) < 0.0f)
		{
			rays[idx].nodeIndex++;
			return;
		}

		if (((pi[0] - triangle->v3[0]) * triangle->n31[0] +
			 (pi[1] - triangle->v3[1]) * triangle->n31[1] +
			 (pi[2] - triangle->v3[2]) * triangle->n31[2]) < 0.0f)
		{
			rays[idx].nodeIndex++;
			return;
		}

		rays[idx].distance = t;

		// TODO
		// What to do?
	}

	rays[idx].nodeIndex++;
}

__device__ int cu_BVHgpu_aabbIntersection(BVHgpu_node *gpu_node, BVHgpu_ray *ray, float *dirInv)
{
	// Calculate intersection
	float tpos[3];
	float tneg[3];
	float tmax[3];
	float tmin[3];
	float lmin;
	float lmax;

	tpos[0] = (gpu_node->max[0] - ray->pos[0]) * dirInv[0];
	tpos[1] = (gpu_node->max[1] - ray->pos[1]) * dirInv[1];
	tpos[2] = (gpu_node->max[2] - ray->pos[2]) * dirInv[2];
	tneg[0] = (gpu_node->min[0] - ray->pos[0]) * dirInv[0];
	tneg[1] = (gpu_node->min[1] - ray->pos[1]) * dirInv[1];
	tneg[2] = (gpu_node->min[2] - ray->pos[2]) * dirInv[2];

	tmax[0] = (tpos[0] > tneg[0]) ? tpos[0] : tneg[0];
	tmax[1] = (tpos[1] > tneg[1]) ? tpos[1] : tneg[1];
	tmax[2] = (tpos[2] > tneg[2]) ? tpos[2] : tneg[2];
	tmin[0] = (tpos[0] < tneg[0]) ? tpos[0] : tneg[0];
	tmin[1] = (tpos[1] < tneg[1]) ? tpos[1] : tneg[1];
	tmin[2] = (tpos[2] < tneg[2]) ? tpos[2] : tneg[2];

	if (tmin[0] >= tmin[1] && tmin[0] >= tmin[2])
		lmin = tmin[0];
	else if (tmin[1] > tmin[0] && tmin[1] >= tmin[2])
		lmin = tmin[1];
	else
		lmin = tmin[2];

	if (tmax[0] <= tmax[1] && tmax[0] <= tmax[2])
		lmax = tmax[0];
	else if (tmax[1] < tmax[0] && tmax[1] <= tmax[2])
		lmax = tmax[1];
	else
		lmax = tmax[2];

	if (ray->distance == -1.0f)
		return lmax > lmin && lmax > 0.0f;
	else
		return lmax > lmin && lmax > 0.0f && lmin < ray->distance;
}

__global__ void cu_BVHgpu_trace(unsigned int gpu_nNodes, BVHgpu_node *gpu_nodes, BVHgpu_ray *rays)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (rays[idx].nodeIndex >= gpu_nNodes)
		return;

	// Normalize ray
	float rayModule;
	float dirInv[3];

	rayModule = sqrtf(
		rays[idx].dir[0] * rays[idx].dir[0] +
		rays[idx].dir[1] * rays[idx].dir[1] +
		rays[idx].dir[2] * rays[idx].dir[2]);
	rays[idx].dir[0] /= rayModule;
	rays[idx].dir[1] /= rayModule;
	rays[idx].dir[2] /= rayModule;

	// Inverse ray
	dirInv[0] = 1.0f / rays[idx].dir[0];
	dirInv[1] = 1.0f / rays[idx].dir[1];
	dirInv[2] = 1.0f / rays[idx].dir[2];

	// Tree traversal
	while (rays[idx].nodeIndex < gpu_nNodes)
	{
		if (cu_BVHgpu_aabbIntersection(&gpu_nodes[rays[idx].nodeIndex], &rays[idx], dirInv))
		{
			if (gpu_nodes[rays[idx].nodeIndex].primitiveLocation != BVHGPU_NOPRIMITIVE)
				return;

			rays[idx].nodeIndex++;
		}
		else
		{
			rays[idx].nodeIndex = gpu_nodes[rays[idx].nodeIndex].skipIndex;
		}
	}
}

inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void cu_BVHgpu_render(
	// Scene definition
	unsigned int gpu_nPrimitives, unsigned int *gpu_primitiveLocations,
	unsigned int gpu_primitiveSize, unsigned char *gpu_primitives,
	unsigned int gpu_nNodes, BVHgpu_node *gpu_nodes,
	// Render buffer
	unsigned int width, unsigned int height, BVH_gpuRenderBuffer *renderBuffer)
{
		/* Variables */

	int pos;
	int nRays;
	BVHgpu_ray *rays;
	int nThreads = 256;
	int nBlocks = 256;
	int nRaysParallel = nThreads * nBlocks;

	unsigned int *cu_gpu_primitiveLocations;
	unsigned char *cu_gpu_primitives;
	BVHgpu_node *cu_gpu_nodes;
	BVHgpu_ray *cu_rays;

	struct timespec time0, time1;
	double timeInit = 0.0;
	double timeAllocation = 0.0;
	double timeHostToDevice = 0.0;
	double timeRaysToDevice = 0.0;
	double timeRaysToHost = 0.0;
	double timeKernel = 0.0;
	double timeOthers = 0.0;


		/* Init rays */

	clock_gettime(CLOCK_REALTIME, &time0);
	nRays = width * height;
	rays = new BVHgpu_ray[nRays];

	for (unsigned int x = 0; x < width; x++)
	{
		for (unsigned int y = 0; y < height; y++)
		{
			pos = x + y * width;

			// Ray position in the screen
			rays[pos].xScreen = x;
			rays[pos].yScreen = y;

			// Init ray position
			rays[pos].pos[0] = 6.0f;
			rays[pos].pos[1] = 3.0f;
			rays[pos].pos[2] = 0.0f;

			// Generate ray direction
			rays[pos].dir[0] = -1.0f;
			rays[pos].dir[1] = ((float)y - 0.5f * (float)height) / (float)width;
			rays[pos].dir[2] = ((float)x - 0.5f * (float)width) / (float)width;

			// Reset distance
			rays[pos].distance = -1.0f;

			// Start from first node
			rays[pos].nodeIndex = 0;

			// Reset buffer
			renderBuffer[x + y * width].rgb[0] = 0.0f;
			renderBuffer[x + y * width].rgb[1] = 0.0f;
			renderBuffer[x + y * width].rgb[2] = 0.0f;
			renderBuffer[x + y * width].alpha = 0.0f;
		}
	}
	clock_gettime(CLOCK_REALTIME, &time1);
	timeInit += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Copy data to GPU memory */

	// Memory allocation
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMalloc((void**)&cu_gpu_primitiveLocations, gpu_nPrimitives * sizeof(unsigned int)));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_primitives, gpu_primitiveSize));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodes, sizeof(BVHgpu_node) * gpu_nNodes));
	cutilSafeCall(hipMalloc((void**)&cu_rays, sizeof(BVHgpu_ray) * nRaysParallel));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeAllocation += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Copy data to graphic card
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMemcpy(cu_gpu_primitiveLocations, gpu_primitiveLocations, gpu_nPrimitives * sizeof(unsigned int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_primitives, gpu_primitives, gpu_primitiveSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodes, gpu_nodes, sizeof(BVHgpu_node) * gpu_nNodes, hipMemcpyHostToDevice));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeHostToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Launch threads */

	dim3 threads(nThreads, 1);
    dim3 grid(nBlocks, 1);

    while (nRays > 0)
    {
    	clock_gettime(CLOCK_REALTIME, &time0);
    	cutilSafeCall(hipMemcpy(cu_rays, rays, sizeof(BVHgpu_ray) * nRaysParallel, hipMemcpyHostToDevice));
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

    	clock_gettime(CLOCK_REALTIME, &time0);
    	cu_BVHgpu_trace<<<grid, threads>>>(gpu_nNodes, cu_gpu_nodes, cu_rays);
    	hipDeviceSynchronize();
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

    	clock_gettime(CLOCK_REALTIME, &time0);
    	cu_BVHgpu_primitiveIntersection<<<grid, threads>>>(gpu_nNodes, cu_gpu_nodes, cu_rays, cu_gpu_primitives);
    	hipDeviceSynchronize();
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

    	clock_gettime(CLOCK_REALTIME, &time0);
    	cutilSafeCall(hipMemcpy(rays, cu_rays, sizeof(BVHgpu_ray) * nRaysParallel, hipMemcpyDeviceToHost));
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

    	clock_gettime(CLOCK_REALTIME, &time0);
    	for (int i = 0; i < nRaysParallel; i++)
    	{
    		if (rays[i].nodeIndex >= gpu_nNodes)
    		{
    			if (rays[i].distance != -1.0f)
				{
					// Fill buffer
					renderBuffer[rays[i].xScreen + rays[i].yScreen * width].rgb[0] = rays[i].distance * 0.03f;
					renderBuffer[rays[i].xScreen + rays[i].yScreen * width].rgb[1] = rays[i].distance * 0.03f;
					renderBuffer[rays[i].xScreen + rays[i].yScreen * width].rgb[2] = rays[i].distance * 0.03f;
					renderBuffer[rays[i].xScreen + rays[i].yScreen * width].alpha = 1.0f;
				}

    			/*
    			if (nRays > nRaysParallel)
    				rays[i] = rays[nRays - 1];

    			nRays--;*/
    		}
    	}

    	for (int i = 0; i < nRaysParallel && i < nRays; i++)
    	{
    		if (rays[i].nodeIndex >= gpu_nNodes)
    		{
    			rays[i] = rays[nRays - 1];
    			nRays--;
    		}
    	}
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeOthers += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    }

    std::cout << "Time init: " << timeInit << std::endl;
    std::cout << "Time allocation: " << timeAllocation << std::endl;
    std::cout << "Time host to device: " << timeHostToDevice << std::endl;
    std::cout << "Time rays to device: " << timeRaysToDevice << std::endl;
    std::cout << "Time rays to host: " << timeRaysToHost << std::endl;
    std::cout << "Time kernel: " << timeKernel << std::endl;
    std::cout << "Time others: " << timeOthers << std::endl;
}
