#include "hip/hip_runtime.h"
/*
 * bvhgpu.cu
 *
 *  Created on: 07 Feb 2011
 *      Author: showroom
 */


/*
 * INCLUDES
 */

#include <cutil_inline.h>

#include "bvhgpu.h"


/*
 * FUNCTIONS
 */

__device__ int cu_BVHgpu_primitiveIntersection(BVHgpu_ray *ray, unsigned char *primitive)
{
	// Variables
	float t;
	float pi[3];
	BVHPrimitive_triangle *triangle;
	BVHgpu_primitiveHeader *header = (BVHgpu_primitiveHeader*)primitive;
	unsigned char *data = primitive + sizeof(BVHgpu_primitiveHeader);

	if (header->type == BVHPRIMITIVE_TYPE_TRIANGLE)
	{
		// Get triangle data
		triangle = (BVHPrimitive_triangle*)data;

		// Calculate t
		t = ((triangle->v1[0] - ray->pos[0]) * triangle->normal[0] +
			 (triangle->v1[1] - ray->pos[1]) * triangle->normal[1] +
			 (triangle->v1[2] - ray->pos[2]) * triangle->normal[2]) /
			(ray->dir[0] * triangle->normal[0] +
			 ray->dir[1] * triangle->normal[1] +
			 ray->dir[2] * triangle->normal[2]);

		// Check direction
		if (t < 0.0f)
			return 0;

		if (ray->distance != -1.0f && t > ray->distance)
			return 0;

		// Check intersection
		pi[0] = ray->pos[0] + ray->dir[0] * t;
		pi[1] = ray->pos[1] + ray->dir[1] * t;
		pi[2] = ray->pos[2] + ray->dir[2] * t;

		if (((pi[0] - triangle->v1[0]) * triangle->n12[0] +
			 (pi[1] - triangle->v1[1]) * triangle->n12[1] +
			 (pi[2] - triangle->v1[2]) * triangle->n12[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v2[0]) * triangle->n23[0] +
			 (pi[1] - triangle->v2[1]) * triangle->n23[1] +
			 (pi[2] - triangle->v2[2]) * triangle->n23[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v3[0]) * triangle->n31[0] +
			 (pi[1] - triangle->v3[1]) * triangle->n31[1] +
			 (pi[2] - triangle->v3[2]) * triangle->n31[2]) < 0.0f)
			return 0;

		ray->distance = t;
		return 1;
	}

	return 0;
}

__device__ int cu_BVHgpu_aabbIntersection(BVHgpu_node *gpu_node, BVHgpu_ray *ray, float *dirInv)
{
	// Calculate intersection
	float tpos[3];
	float tneg[3];
	float tmax[3];
	float tmin[3];
	float lmin;
	float lmax;

	tpos[0] = (gpu_node->max[0] - ray->pos[0]) * dirInv[0];
	tpos[1] = (gpu_node->max[1] - ray->pos[1]) * dirInv[1];
	tpos[2] = (gpu_node->max[2] - ray->pos[2]) * dirInv[2];
	tneg[0] = (gpu_node->min[0] - ray->pos[0]) * dirInv[0];
	tneg[1] = (gpu_node->min[1] - ray->pos[1]) * dirInv[1];
	tneg[2] = (gpu_node->min[2] - ray->pos[2]) * dirInv[2];

	tmax[0] = (tpos[0] > tneg[0]) ? tpos[0] : tneg[0];
	tmax[1] = (tpos[1] > tneg[1]) ? tpos[1] : tneg[1];
	tmax[2] = (tpos[2] > tneg[2]) ? tpos[2] : tneg[2];
	tmin[0] = (tpos[0] < tneg[0]) ? tpos[0] : tneg[0];
	tmin[1] = (tpos[1] < tneg[1]) ? tpos[1] : tneg[1];
	tmin[2] = (tpos[2] < tneg[2]) ? tpos[2] : tneg[2];

	if (tmin[0] >= tmin[1] && tmin[0] >= tmin[2])
		lmin = tmin[0];
	else if (tmin[1] > tmin[0] && tmin[1] >= tmin[2])
		lmin = tmin[1];
	else
		lmin = tmin[2];

	if (tmax[0] <= tmax[1] && tmax[0] <= tmax[2])
		lmax = tmax[0];
	else if (tmax[1] < tmax[0] && tmax[1] <= tmax[2])
		lmax = tmax[1];
	else
		lmax = tmax[2];

	if (ray->distance == -1.0f)
		return lmax > lmin && lmax > 0.0f;
	else
		return lmax > lmin && lmax > 0.0f && lmin < ray->distance;
}

__global__ void cu_BVHgpu_trace(const unsigned int gpu_nNodes, BVHgpu_node *gpu_nodes, BVHgpu_ray *rays, unsigned char *primitives)
{
	// Normalize ray
	float rayModule;
	float dirInv[3];
	unsigned int nodeIndex = 0;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	rayModule = sqrtf(
		rays[idx].dir[0] * rays[idx].dir[0] +
		rays[idx].dir[1] * rays[idx].dir[1] +
		rays[idx].dir[2] * rays[idx].dir[2]);
	rays[idx].dir[0] /= rayModule;
	rays[idx].dir[1] /= rayModule;
	rays[idx].dir[2] /= rayModule;

	// Inverse ray
	dirInv[0] = 1.0f / rays[idx].dir[0];
	dirInv[1] = 1.0f / rays[idx].dir[1];
	dirInv[2] = 1.0f / rays[idx].dir[2];

	// Tree traversal
	while (nodeIndex < gpu_nNodes)
	{
		if (cu_BVHgpu_aabbIntersection(&gpu_nodes[nodeIndex], &rays[idx], dirInv))
		{
			if (gpu_nodes[nodeIndex].primitiveLocation != BVHGPU_NOPRIMITIVE)
			{
				if (cu_BVHgpu_primitiveIntersection(&rays[idx], &primitives[gpu_nodes[nodeIndex].primitiveLocation]))
					rays[idx].closestNode = nodeIndex;
			}

			nodeIndex++;
		}
		else
		{
			nodeIndex = gpu_nodes[nodeIndex].skipIndex;
		}
	}
}

inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void cu_BVHgpu_render(
	// Scene definition
	unsigned int gpu_nPrimitives, unsigned int *gpu_primitiveLocations,
	unsigned int gpu_primitiveSize, unsigned char *gpu_primitives,
	unsigned int gpu_nNodes, BVHgpu_node *gpu_nodes,
	// Render buffer
	unsigned int width, unsigned int height, BVH_gpuRenderBuffer *renderBuffer)
{
		/* Variables */

	int pos;
	int nRays;
	BVHgpu_ray *rays;
	int nThreads = 256;
	int nBlocks = 256;
	int nRaysParallel = nThreads * nBlocks;

	unsigned int *cu_gpu_primitiveLocations;
	unsigned char *cu_gpu_primitives;
	BVHgpu_node *cu_gpu_nodes;
	BVHgpu_ray *cu_rays;

	struct timespec time0, time1;
	double timeInit = 0.0;
	double timeAllocation = 0.0;
	double timeHostToDevice = 0.0;
	double timeRaysToDevice = 0.0;
	double timeRaysToHost = 0.0;
	double timeKernel = 0.0;
	double timeOthers = 0.0;
	double timeCPU;
	double timeTotal;
	double timeMemory;


		/* Init rays */

	clock_gettime(CLOCK_REALTIME, &time0);
	nRays = width * height;
	rays = new BVHgpu_ray[nRays];

	for (unsigned int x = 0; x < width; x++)
	{
		for (unsigned int y = 0; y < height; y++)
		{
			pos = x + y * width;

			// Init ray position
			rays[pos].pos[0] = 6.0f;
			rays[pos].pos[1] = 3.0f;
			rays[pos].pos[2] = 0.0f;

			// Generate ray direction
			rays[pos].dir[0] = -1.0f;
			rays[pos].dir[1] = ((float)y - 0.5f * (float)height) / (float)width;
			rays[pos].dir[2] = ((float)x - 0.5f * (float)width) / (float)width;

			/*
			// Init ray position
			rays[pos].pos[0] = 1.0f;
			rays[pos].pos[1] = 11.0f;
			rays[pos].pos[2] = 5.0f;

			// Generate ray direction
			rays[pos].dir[0] = 1.0f;
			rays[pos].dir[1] = ((float)x - 0.5f * (float)height) / (float)width;
			rays[pos].dir[2] = ((float)y - 0.5f * (float)width) / (float)width;*/

			// Reset distance
			rays[pos].distance = -1.0f;

			// Reset buffer
			renderBuffer[pos].rgb[0] = 0.0f;
			renderBuffer[pos].rgb[1] = 0.0f;
			renderBuffer[pos].rgb[2] = 0.0f;
			renderBuffer[pos].alpha = 0.0f;
		}
	}
	clock_gettime(CLOCK_REALTIME, &time1);
	timeInit += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Copy data to GPU memory */

	// Memory allocation
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMalloc((void**)&cu_gpu_primitiveLocations, gpu_nPrimitives * sizeof(unsigned int)));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_primitives, gpu_primitiveSize));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodes, sizeof(BVHgpu_node) * gpu_nNodes));
	cutilSafeCall(hipMalloc((void**)&cu_rays, sizeof(BVHgpu_ray) * nRays));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeAllocation += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Copy data to graphic card
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMemcpy(cu_gpu_primitiveLocations, gpu_primitiveLocations, gpu_nPrimitives * sizeof(unsigned int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_primitives, gpu_primitives, gpu_primitiveSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodes, gpu_nodes, sizeof(BVHgpu_node) * gpu_nNodes, hipMemcpyHostToDevice));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeHostToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Launch threads */

	dim3 threads(nThreads, 1);
    dim3 grid(nBlocks, 1);

    for (int i = 0; i < nRays; i += nRaysParallel)
    {
    	if ((nRays - i) < nRaysParallel)
    	{
    		clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(cu_rays, &rays[i], sizeof(BVHgpu_ray) * (nRays - i), hipMemcpyHostToDevice));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    	}
    	else
    	{
			clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(cu_rays, &rays[i], sizeof(BVHgpu_ray) * nRaysParallel, hipMemcpyHostToDevice));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    	}

    	clock_gettime(CLOCK_REALTIME, &time0);
    	cu_BVHgpu_trace<<<grid, threads>>>(gpu_nNodes, cu_gpu_nodes, cu_rays, cu_gpu_primitives);
    	hipDeviceSynchronize();
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

    	if ((nRays - i) < nRaysParallel)
    	{
    		clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(&rays[i], cu_rays, sizeof(BVHgpu_ray) * (nRays - i), hipMemcpyDeviceToHost));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    	}
    	else
    	{
			clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(&rays[i], cu_rays, sizeof(BVHgpu_ray) * nRaysParallel, hipMemcpyDeviceToHost));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    	}

    	clock_gettime(CLOCK_REALTIME, &time0);
    	for (int j = i; j < nRays && j < i + nRaysParallel; j++)
    	{
			if (rays[j].distance != -1.0f)
			{
				renderBuffer[j].rgb[0] = rays[j].distance * 0.03f;
				renderBuffer[j].rgb[1] = rays[j].distance * 0.03f;
				renderBuffer[j].rgb[2] = rays[j].distance * 0.03f;
				renderBuffer[j].alpha = 1.0f;
			}
    	}
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeOthers += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    }

    timeMemory = timeAllocation + timeHostToDevice + timeRaysToDevice + timeRaysToHost;
    timeCPU = timeInit + timeOthers;
    timeTotal = timeKernel + timeCPU + timeMemory;

    std::cout << "Time total             : " << timeTotal << std::endl;
    std::cout << "  Time CPU             : " << timeCPU << "\t(" << (100.0 * timeCPU / timeTotal) << "%)" << std::endl;
    std::cout << "    Time init          : " << timeInit << "\t(" << (100.0 * timeInit / timeTotal) << "%)" << std::endl;
    std::cout << "    Time others        : " << timeOthers << "\t(" << (100.0 * timeOthers / timeTotal) << "%)" << std::endl;
    std::cout << "  Time memory          : " << timeMemory << "\t(" << (100.0 * timeMemory / timeTotal) << "%)" << std::endl;
    std::cout << "    Time allocation    : " << timeAllocation << "\t(" << (100.0 * timeAllocation / timeTotal) << "%)" << std::endl;
    std::cout << "    Time host to device: " << timeHostToDevice << "\t(" << (100.0 * timeHostToDevice / timeTotal) << "%)" << std::endl;
    std::cout << "    Time rays to device: " << timeRaysToDevice << "\t(" << (100.0 * timeRaysToDevice / timeTotal) << "%)" << std::endl;
    std::cout << "    Time rays to host  : " << timeRaysToHost << "\t(" << (100.0 * timeRaysToHost / timeTotal) << "%)" << std::endl;
    std::cout << "  Time Kernel          : " << timeKernel << "\t(" << (100.0 * timeKernel / timeTotal) << "%)" << std::endl;
}
