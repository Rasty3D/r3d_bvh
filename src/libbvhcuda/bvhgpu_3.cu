#include "hip/hip_runtime.h"
/*
 * bvhgpu.cu
 *
 *  Created on: 07 Feb 2011
 *      Author: showroom
 */


/*
 * INCLUDES
 */

#include <cutil_inline.h>

#include "bvhgpu.h"


/*
 * TEXTURES
 */

texture<float4, 1, hipReadModeElementType> nodeTex_min;
texture<float4, 1, hipReadModeElementType> nodeTex_max;
texture<uint2, 1, hipReadModeElementType> nodeTex_skipLocation;


/*
 * FUNCTIONS
 */

__device__ int cu_BVHgpu_primitiveIntersection(BVHgpu_ray *ray, unsigned char *primitive)
{
	// Variables
	float t;
	float pi[3];
	BVHPrimitive_triangle *triangle;
	BVHgpu_primitiveHeader *header = (BVHgpu_primitiveHeader*)primitive;
	unsigned char *data = primitive + sizeof(BVHgpu_primitiveHeader);

	if (header->type == BVHPRIMITIVE_TYPE_TRIANGLE)
	{
		// Get triangle data
		triangle = (BVHPrimitive_triangle*)data;

		// Calculate t
		t = ((triangle->v1[0] - ray->pos[0]) * triangle->normal[0] +
			 (triangle->v1[1] - ray->pos[1]) * triangle->normal[1] +
			 (triangle->v1[2] - ray->pos[2]) * triangle->normal[2]) /
			(ray->dir[0] * triangle->normal[0] +
			 ray->dir[1] * triangle->normal[1] +
			 ray->dir[2] * triangle->normal[2]);

		// Check direction
		if (t < 0.0f)
			return 0;

		if (ray->distance != -1.0f && t > ray->distance)
			return 0;

		// Check intersection
		pi[0] = ray->pos[0] + ray->dir[0] * t;
		pi[1] = ray->pos[1] + ray->dir[1] * t;
		pi[2] = ray->pos[2] + ray->dir[2] * t;

		if (((pi[0] - triangle->v1[0]) * triangle->n12[0] +
			 (pi[1] - triangle->v1[1]) * triangle->n12[1] +
			 (pi[2] - triangle->v1[2]) * triangle->n12[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v2[0]) * triangle->n23[0] +
			 (pi[1] - triangle->v2[1]) * triangle->n23[1] +
			 (pi[2] - triangle->v2[2]) * triangle->n23[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v3[0]) * triangle->n31[0] +
			 (pi[1] - triangle->v3[1]) * triangle->n31[1] +
			 (pi[2] - triangle->v3[2]) * triangle->n31[2]) < 0.0f)
			return 0;

		ray->distance = t;
		return 1;
	}

	return 0;
}

__device__ int cu_BVHgpu_aabbIntersection(int nodeIndex, BVHgpu_ray *ray, float *dirInv)
{
	// Calculate intersection
	float tpos[3];
	float tneg[3];
	float tmax[3];
	float tmin[3];
	float lmin;
	float lmax;
	float4 aabbmin = tex1Dfetch(nodeTex_min, nodeIndex);
	float4 aabbmax = tex1Dfetch(nodeTex_max, nodeIndex);

	tpos[0] = (aabbmax.x - ray->pos[0]) * dirInv[0];
	tpos[1] = (aabbmax.y - ray->pos[1]) * dirInv[1];
	tpos[2] = (aabbmax.z - ray->pos[2]) * dirInv[2];
	tneg[0] = (aabbmin.x - ray->pos[0]) * dirInv[0];
	tneg[1] = (aabbmin.y - ray->pos[1]) * dirInv[1];
	tneg[2] = (aabbmin.z - ray->pos[2]) * dirInv[2];

	tmax[0] = (tpos[0] > tneg[0]) ? tpos[0] : tneg[0];
	tmax[1] = (tpos[1] > tneg[1]) ? tpos[1] : tneg[1];
	tmax[2] = (tpos[2] > tneg[2]) ? tpos[2] : tneg[2];
	tmin[0] = (tpos[0] < tneg[0]) ? tpos[0] : tneg[0];
	tmin[1] = (tpos[1] < tneg[1]) ? tpos[1] : tneg[1];
	tmin[2] = (tpos[2] < tneg[2]) ? tpos[2] : tneg[2];

	if (tmin[0] >= tmin[1] && tmin[0] >= tmin[2])
		lmin = tmin[0];
	else if (tmin[1] > tmin[0] && tmin[1] >= tmin[2])
		lmin = tmin[1];
	else
		lmin = tmin[2];

	if (tmax[0] <= tmax[1] && tmax[0] <= tmax[2])
		lmax = tmax[0];
	else if (tmax[1] < tmax[0] && tmax[1] <= tmax[2])
		lmax = tmax[1];
	else
		lmax = tmax[2];

	if (ray->distance == -1.0f)
		return lmax > lmin && lmax > 0.0f;
	else
		return lmax > lmin && lmax > 0.0f && lmin < ray->distance;
}

__global__ void cu_BVHgpu_trace(const unsigned int gpu_nNodes, BVHgpu_ray *rays, unsigned char *primitives)
{
	// Normalize ray
	float rayModule;
	float dirInv[3];
	unsigned int nodeIndex = 0;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	uint2 skipLocation;

	rayModule = sqrtf(
		rays[idx].dir[0] * rays[idx].dir[0] +
		rays[idx].dir[1] * rays[idx].dir[1] +
		rays[idx].dir[2] * rays[idx].dir[2]);
	rays[idx].dir[0] /= rayModule;
	rays[idx].dir[1] /= rayModule;
	rays[idx].dir[2] /= rayModule;

	// Inverse ray
	dirInv[0] = 1.0f / rays[idx].dir[0];
	dirInv[1] = 1.0f / rays[idx].dir[1];
	dirInv[2] = 1.0f / rays[idx].dir[2];

	// Tree traversal
	while (nodeIndex < gpu_nNodes)
	{
		skipLocation = tex1Dfetch(nodeTex_skipLocation, nodeIndex);

		if (cu_BVHgpu_aabbIntersection(nodeIndex, &rays[idx], dirInv))
		{
			if (skipLocation.y != BVHGPU_NOPRIMITIVE)
			{
				if (cu_BVHgpu_primitiveIntersection(&rays[idx], &primitives[skipLocation.y]))
					rays[idx].closestNode = nodeIndex;
			}

			nodeIndex++;
		}
		else
		{
			nodeIndex = skipLocation.x;
		}
	}
}

inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void cu_BVHgpu_render(
	// Scene definition
	unsigned int gpu_nPrimitives, unsigned int *gpu_primitiveLocations,
	unsigned int gpu_primitiveSize, unsigned char *gpu_primitives,
	unsigned int gpu_nNodes, float *gpu_nodesMin, float *gpu_nodesMax, unsigned int *gpu_nodesSkipLocation,
	// Render buffer
	unsigned int width, unsigned int height, BVH_gpuRenderBuffer *renderBuffer,
	// Kernel control
	int nThreads, int nBlocks)
{
		/* Variables */

	int pos;
	int nRays;
	BVHgpu_ray *rays;
	int nRaysParallel = nThreads * nBlocks;
	dim3 threads(nThreads, 1);
    dim3 grid(nBlocks, 1);

	unsigned char *cu_gpu_primitives;
	BVHgpu_ray *cu_rays;
	float *cu_gpu_nodesMin;
	float *cu_gpu_nodesMax;
	unsigned int *cu_gpu_nodesSkipLocation;

	struct timespec time0, time1;
	double timeInit = 0.0;
	double timeAllocation = 0.0;
	double timeHostToDevice = 0.0;
	double timeRaysToDevice = 0.0;
	double timeRaysToHost = 0.0;
	double timeKernel = 0.0;
	double timeOthers = 0.0;
	double timeBindTextures = 0.0;
	double timeCPU;
	double timeTotal;
	double timeMemory;


		/* Configure textures */

	// Min
	nodeTex_min.normalized = 0;
	nodeTex_min.filterMode = hipFilterModePoint;
	nodeTex_min.addressMode[0] = hipAddressModeClamp;

	// Max
	nodeTex_max.normalized = 0;
	nodeTex_max.filterMode = hipFilterModePoint;
	nodeTex_max.addressMode[0] = hipAddressModeClamp;

	// Skip index and primitive location
	nodeTex_skipLocation.normalized = 0;
	nodeTex_skipLocation.filterMode = hipFilterModePoint;
	nodeTex_skipLocation.addressMode[0] = hipAddressModeClamp;


		/* Init rays */

	clock_gettime(CLOCK_REALTIME, &time0);
	nRays = width * height;
	rays = new BVHgpu_ray[nRays];

	for (unsigned int x = 0; x < width; x++)
	{
		for (unsigned int y = 0; y < height; y++)
		{
			pos = x + y * width;

			// Init ray position
			rays[pos].pos[0] = 6.0f;
			rays[pos].pos[1] = 3.0f;
			rays[pos].pos[2] = 0.0f;

			// Generate ray direction
			rays[pos].dir[0] = -1.0f;
			rays[pos].dir[1] = ((float)y - 0.5f * (float)height) / (float)width;
			rays[pos].dir[2] = ((float)x - 0.5f * (float)width) / (float)width;

			/*
			// Init ray position
			rays[pos].pos[0] = 1.0f;
			rays[pos].pos[1] = 11.0f;
			rays[pos].pos[2] = 5.0f;

			// Generate ray direction
			rays[pos].dir[0] = 1.0f;
			rays[pos].dir[1] = ((float)x - 0.5f * (float)height) / (float)width;
			rays[pos].dir[2] = ((float)y - 0.5f * (float)width) / (float)width;*/

			// Reset distance
			rays[pos].distance = -1.0f;

			// Reset buffer
			renderBuffer[pos].rgb[0] = 0.0f;
			renderBuffer[pos].rgb[1] = 0.0f;
			renderBuffer[pos].rgb[2] = 0.0f;
			renderBuffer[pos].alpha = 0.0f;
		}
	}
	clock_gettime(CLOCK_REALTIME, &time1);
	timeInit += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Copy data to GPU memory */

	// Memory allocation
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMalloc((void**)&cu_gpu_primitives, gpu_primitiveSize));
	cutilSafeCall(hipMalloc((void**)&cu_rays, sizeof(BVHgpu_ray) * nRays));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodesMin, gpu_nNodes * 4 * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodesMax, gpu_nNodes * 4 * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodesSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int)));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeAllocation += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Copy data to graphic card
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMemcpy(cu_gpu_primitives, gpu_primitives, gpu_primitiveSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodesMin, gpu_nodesMin, gpu_nNodes * 4 * sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodesMax, gpu_nodesMax, gpu_nNodes * 4 * sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodesSkipLocation, gpu_nodesSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int), hipMemcpyHostToDevice));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeHostToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Bind data to textures
	clock_gettime(CLOCK_REALTIME, &time0);
	hipChannelFormatDesc channelDescMin = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, nodeTex_min, cu_gpu_nodesMin, channelDescMin, gpu_nNodes * 4 * sizeof(float));
	hipChannelFormatDesc channelDescMax = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, nodeTex_max, cu_gpu_nodesMax, channelDescMax, gpu_nNodes * 4 * sizeof(float));
	hipChannelFormatDesc channelDescSkipLocation = hipCreateChannelDesc<uint2>();
	hipBindTexture(NULL, nodeTex_skipLocation, cu_gpu_nodesSkipLocation, channelDescSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeBindTextures += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Launch threads */

    for (int i = 0; i < nRays; i += nRaysParallel)
    {
    	if ((nRays - i) < nRaysParallel)
    	{
    		clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(cu_rays, &rays[i], sizeof(BVHgpu_ray) * (nRays - i), hipMemcpyHostToDevice));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
			grid.x = iDivUp(nRays - i, nThreads);
    	}
    	else
    	{
			clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(cu_rays, &rays[i], sizeof(BVHgpu_ray) * nRaysParallel, hipMemcpyHostToDevice));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
			grid.x = nBlocks;
    	}

    	clock_gettime(CLOCK_REALTIME, &time0);
    	cu_BVHgpu_trace<<<grid, threads>>>(gpu_nNodes, cu_rays, cu_gpu_primitives);
    	hipDeviceSynchronize();
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

    	if ((nRays - i) < nRaysParallel)
    	{
    		clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(&rays[i], cu_rays, sizeof(BVHgpu_ray) * (nRays - i), hipMemcpyDeviceToHost));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    	}
    	else
    	{
			clock_gettime(CLOCK_REALTIME, &time0);
			cutilSafeCall(hipMemcpy(&rays[i], cu_rays, sizeof(BVHgpu_ray) * nRaysParallel, hipMemcpyDeviceToHost));
			clock_gettime(CLOCK_REALTIME, &time1);
			timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    	}

    	clock_gettime(CLOCK_REALTIME, &time0);
    	for (int j = i; j < nRays && j < i + nRaysParallel; j++)
    	{
			if (rays[j].distance != -1.0f)
			{
				renderBuffer[j].rgb[0] = rays[j].distance * 0.03f;
				renderBuffer[j].rgb[1] = rays[j].distance * 0.03f;
				renderBuffer[j].rgb[2] = rays[j].distance * 0.03f;
				renderBuffer[j].alpha = 1.0f;
			}
    	}
    	clock_gettime(CLOCK_REALTIME, &time1);
    	timeOthers += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
    }

    timeMemory = timeAllocation + timeHostToDevice + timeBindTextures + timeRaysToDevice + timeRaysToHost;
    timeCPU = timeInit + timeOthers;
    timeTotal = timeKernel + timeCPU + timeMemory;

    std::cout << "Time total             : " << timeTotal << std::endl;
    std::cout << "  Time CPU             : " << timeCPU << "\t(" << (100.0 * timeCPU / timeTotal) << "%)" << std::endl;
    std::cout << "    Time init          : " << timeInit << "\t(" << (100.0 * timeInit / timeTotal) << "%)" << std::endl;
    std::cout << "    Time others        : " << timeOthers << "\t(" << (100.0 * timeOthers / timeTotal) << "%)" << std::endl;
    std::cout << "  Time memory          : " << timeMemory << "\t(" << (100.0 * timeMemory / timeTotal) << "%)" << std::endl;
    std::cout << "    Time allocation    : " << timeAllocation << "\t(" << (100.0 * timeAllocation / timeTotal) << "%)" << std::endl;
    std::cout << "    Time host to device: " << timeHostToDevice << "\t(" << (100.0 * timeHostToDevice / timeTotal) << "%)" << std::endl;
    std::cout << "    Time bind textures : " << timeBindTextures  << "\t(" << (100.0 * timeBindTextures / timeTotal) << "%)" << std::endl;
    std::cout << "    Time rays to device: " << timeRaysToDevice << "\t(" << (100.0 * timeRaysToDevice / timeTotal) << "%)" << std::endl;
    std::cout << "    Time rays to host  : " << timeRaysToHost << "\t(" << (100.0 * timeRaysToHost / timeTotal) << "%)" << std::endl;
    std::cout << "  Time Kernel          : " << timeKernel << "\t(" << (100.0 * timeKernel / timeTotal) << "%)" << std::endl;


    	/* Free memory */

    cutilSafeCall(hipFree(cu_gpu_nodesMin));
    cutilSafeCall(hipFree(cu_gpu_nodesMax));
    cutilSafeCall(hipFree(cu_gpu_nodesSkipLocation));
	cutilSafeCall(hipFree(cu_gpu_primitives));
	cutilSafeCall(hipFree(cu_rays));
}
