#include "hip/hip_runtime.h"
/*
 * bvhgpu.cu
 *
 *  Created on: 07 Feb 2011
 *      Author: showroom
 */


/*
 * INCLUDES
 */

#include <cutil_inline.h>

#include "bvhgpu.h"


/*
 * TEXTURES
 */

texture<float4, 1, hipReadModeElementType> nodeTex_min;
texture<float4, 1, hipReadModeElementType> nodeTex_max;
texture<uint2, 1, hipReadModeElementType> nodeTex_skipLocation;


/*
 * FUNCTIONS
 */

__device__ int cu_BVHgpu_primitiveIntersection(BVHgpu_ray *ray, unsigned char *primitive)
{
	// Variables
	float t;
	float pi[3];
	BVHPrimitive_triangle *triangle;
	BVHgpu_primitiveHeader *header = (BVHgpu_primitiveHeader*)primitive;
	unsigned char *data = primitive + sizeof(BVHgpu_primitiveHeader);

	if (header->type == BVHPRIMITIVE_TYPE_TRIANGLE)
	{
		// Get triangle data
		triangle = (BVHPrimitive_triangle*)data;

		// Calculate t
		t = ((triangle->v1[0] - ray->pos[0]) * triangle->normal[0] +
			 (triangle->v1[1] - ray->pos[1]) * triangle->normal[1] +
			 (triangle->v1[2] - ray->pos[2]) * triangle->normal[2]) /
			(ray->dir[0] * triangle->normal[0] +
			 ray->dir[1] * triangle->normal[1] +
			 ray->dir[2] * triangle->normal[2]);

		// Check direction
		if (t < 0.0f)
			return 0;

		if (ray->distance != -1.0f && t > ray->distance)
			return 0;

		// Check intersection
		pi[0] = ray->pos[0] + ray->dir[0] * t;
		pi[1] = ray->pos[1] + ray->dir[1] * t;
		pi[2] = ray->pos[2] + ray->dir[2] * t;

		if (((pi[0] - triangle->v1[0]) * triangle->n12[0] +
			 (pi[1] - triangle->v1[1]) * triangle->n12[1] +
			 (pi[2] - triangle->v1[2]) * triangle->n12[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v2[0]) * triangle->n23[0] +
			 (pi[1] - triangle->v2[1]) * triangle->n23[1] +
			 (pi[2] - triangle->v2[2]) * triangle->n23[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v3[0]) * triangle->n31[0] +
			 (pi[1] - triangle->v3[1]) * triangle->n31[1] +
			 (pi[2] - triangle->v3[2]) * triangle->n31[2]) < 0.0f)
			return 0;

		ray->distance = t;
		return 1;
	}

	return 0;
}

__device__ int cu_BVHgpu_aabbIntersection(int nodeIndex, BVHgpu_ray *ray, float *dirInv)
{
	// Calculate intersection
	float tpos[3];
	float tneg[3];
	float tmax[3];
	float tmin[3];
	float lmin;
	float lmax;
	float4 aabbmin = tex1Dfetch(nodeTex_min, nodeIndex);
	float4 aabbmax = tex1Dfetch(nodeTex_max, nodeIndex);

	tpos[0] = (aabbmax.x - ray->pos[0]) * dirInv[0];
	tpos[1] = (aabbmax.y - ray->pos[1]) * dirInv[1];
	tpos[2] = (aabbmax.z - ray->pos[2]) * dirInv[2];
	tneg[0] = (aabbmin.x - ray->pos[0]) * dirInv[0];
	tneg[1] = (aabbmin.y - ray->pos[1]) * dirInv[1];
	tneg[2] = (aabbmin.z - ray->pos[2]) * dirInv[2];

	tmax[0] = max(tpos[0], tneg[0]);
	tmax[1] = max(tpos[1], tneg[1]);
	tmax[2] = max(tpos[2], tneg[2]);
	tmin[0] = min(tpos[0], tneg[0]);
	tmin[1] = min(tpos[1], tneg[1]);
	tmin[2] = min(tpos[2], tneg[2]);

	lmin = max(tmin[0], max(tmin[1], tmin[2]));
	lmax = min(tmax[0], min(tmax[1], tmax[2]));

	if (ray->distance == -1.0f)
		return lmax > lmin && lmax > 0.0f;
	else
		return lmax > lmin && lmax > 0.0f && lmin < ray->distance;
}

__global__ void cu_BVHgpu_trace(const unsigned int gpu_nNodes, BVHgpu_ray *rays, const int nRays, unsigned char *primitives)
{
	// Normalize ray
	float rayModule;
	float dirInv[3];
	unsigned int nodeIndex = 0;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	uint2 skipLocation;

	if (idx >= nRays)
		return;

	rayModule = sqrtf(
		rays[idx].dir[0] * rays[idx].dir[0] +
		rays[idx].dir[1] * rays[idx].dir[1] +
		rays[idx].dir[2] * rays[idx].dir[2]);
	rays[idx].dir[0] /= rayModule;
	rays[idx].dir[1] /= rayModule;
	rays[idx].dir[2] /= rayModule;

	// Inverse ray
	dirInv[0] = 1.0f / rays[idx].dir[0];
	dirInv[1] = 1.0f / rays[idx].dir[1];
	dirInv[2] = 1.0f / rays[idx].dir[2];

	// Tree traversal
	while (nodeIndex < gpu_nNodes)
	{
		skipLocation = tex1Dfetch(nodeTex_skipLocation, nodeIndex);

		if (cu_BVHgpu_aabbIntersection(nodeIndex, &rays[idx], dirInv))
		{
			if (skipLocation.y != BVHGPU_NOPRIMITIVE)
			{
				if (cu_BVHgpu_primitiveIntersection(&rays[idx], &primitives[skipLocation.y]))
					rays[idx].closestNode = nodeIndex;
			}

			nodeIndex++;
		}
		else
		{
			nodeIndex = skipLocation.x;
		}
	}
}

inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

void cu_BVHgpu_render(
	// Scene definition
	unsigned int gpu_nPrimitives, unsigned int *gpu_primitiveLocations,
	unsigned int gpu_primitiveSize, unsigned char *gpu_primitives,
	unsigned int gpu_nNodes, float *gpu_nodesMin, float *gpu_nodesMax, unsigned int *gpu_nodesSkipLocation,
	// Render buffer
	unsigned int width, unsigned int height, BVH_gpuRenderBuffer *renderBuffer, BVHCamera &camera,
	// Kernel control
	int nThreads, int nBlocks)
{
		/* Variables */

	int pos;
	int nRays;
	int nRaysF[2];	// Fixed number of rays (for the last iteration)
	BVHgpu_ray *rays[2];
	int rayBuffer = 0;
	dim3 threads(nThreads, 1);
	dim3 grid(nBlocks, 1);
	int x, y;
	int k2;

	unsigned char *cu_gpu_primitives;
	BVHgpu_ray *cu_rays;
	float *cu_gpu_nodesMin;
	float *cu_gpu_nodesMax;
	unsigned int *cu_gpu_nodesSkipLocation;

	int kernelDim[2] = {3, 3};
	int kernelIdx[2] = {0, 0};
	int kernelSize = kernelDim[0] * kernelDim[1];
	int nRaysTotal = width * height * kernelSize;

	struct timespec time0, time1;
	double timeInit = 0.0;
	double timeAllocation = 0.0;
	double timeHostToDevice = 0.0;
	double timeRaysToDevice = 0.0;
	double timeRaysToHost = 0.0;
	double timeKernel = 0.0;
	double timeOthers = 0.0;
	double timeBindTextures = 0.0;
	double timeGenRays = 0.0;
	double timeFillBuffer = 0.0;
	double timeCPU;
	double timeTotal;
	double timeMemory;


		/* Configure textures */

	// Min
	nodeTex_min.normalized = 0;
	nodeTex_min.filterMode = hipFilterModePoint;
	nodeTex_min.addressMode[0] = hipAddressModeClamp;

	// Max
	nodeTex_max.normalized = 0;
	nodeTex_max.filterMode = hipFilterModePoint;
	nodeTex_max.addressMode[0] = hipAddressModeClamp;

	// Skip index and primitive location
	nodeTex_skipLocation.normalized = 0;
	nodeTex_skipLocation.filterMode = hipFilterModePoint;
	nodeTex_skipLocation.addressMode[0] = hipAddressModeClamp;


		/* Init render buffer */

	clock_gettime(CLOCK_REALTIME, &time0);
	nRays = nThreads * nBlocks;
	nRaysF[0] = nRays;
	nRaysF[1] = nRays;
	rays[0] = new BVHgpu_ray[nRays];
	rays[1] = new BVHgpu_ray[nRays];

	for (unsigned int x = 0; x < width; x++)
	{
		for (unsigned int y = 0; y < height; y++)
		{
			pos = x + y * width;

			// Reset buffer
			renderBuffer[pos].rgb[0] = 0.0f;
			renderBuffer[pos].rgb[1] = 0.0f;
			renderBuffer[pos].rgb[2] = 0.0f;
			renderBuffer[pos].alpha = 0.0f;
		}
	}
	clock_gettime(CLOCK_REALTIME, &time1);
	timeInit += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Copy data to GPU memory */

	// Memory allocation
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMalloc((void**)&cu_gpu_primitives, gpu_primitiveSize));
	cutilSafeCall(hipMalloc((void**)&cu_rays, sizeof(BVHgpu_ray) * nRays));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodesMin, gpu_nNodes * 4 * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodesMax, gpu_nNodes * 4 * sizeof(float)));
	cutilSafeCall(hipMalloc((void**)&cu_gpu_nodesSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int)));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeAllocation += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Copy data to graphic card
	clock_gettime(CLOCK_REALTIME, &time0);
	cutilSafeCall(hipMemcpy(cu_gpu_primitives, gpu_primitives, gpu_primitiveSize, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodesMin, gpu_nodesMin, gpu_nNodes * 4 * sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodesMax, gpu_nodesMax, gpu_nNodes * 4 * sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(cu_gpu_nodesSkipLocation, gpu_nodesSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int), hipMemcpyHostToDevice));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeHostToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Bind data to textures
	clock_gettime(CLOCK_REALTIME, &time0);
	hipChannelFormatDesc channelDescMin = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, nodeTex_min, cu_gpu_nodesMin, channelDescMin, gpu_nNodes * 4 * sizeof(float));
	hipChannelFormatDesc channelDescMax = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, nodeTex_max, cu_gpu_nodesMax, channelDescMax, gpu_nNodes * 4 * sizeof(float));
	hipChannelFormatDesc channelDescSkipLocation = hipCreateChannelDesc<uint2>();
	hipBindTexture(NULL, nodeTex_skipLocation, cu_gpu_nodesSkipLocation, channelDescSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int));
	clock_gettime(CLOCK_REALTIME, &time1);
	timeBindTextures += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Do the render */

	// Generate first rays
	clock_gettime(CLOCK_REALTIME, &time0);

	if (nRaysTotal < nRays)
		nRaysF[0] = nRaysTotal;

	for (unsigned int i = 0; i < nRaysF[0]; i++)
	{
		// Calculate buffer coordinates
		x = (i / kernelSize) % width;
		y = (i / kernelSize) / width;

		// Calculate kernel coordinates
		kernelIdx[0] = (i % kernelSize) % kernelDim[0];
		kernelIdx[1] = (i % kernelSize) / kernelDim[0];

		// Get camera ray
		camera.getRay(
			((float)x * 2.0f - (float)width) / (float)width,
			((float)y * 2.0f - (float)height) / (float)width,
			kernelDim, kernelIdx,
			rays[0][i].pos, rays[0][i].dir);

		// Reset distance
		rays[0][i].distance = -1.0f;
	}
	clock_gettime(CLOCK_REALTIME, &time1);
	timeGenRays += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	rayBuffer = 1;

	for (int k = 0; k < nRaysTotal; k += nRays)
	{
		// Flip buffers
		rayBuffer = !rayBuffer;

		// Copy rays to GPU
		clock_gettime(CLOCK_REALTIME, &time0);
		cutilSafeCall(hipMemcpy(cu_rays, rays[rayBuffer], sizeof(BVHgpu_ray) * nRaysF[rayBuffer], hipMemcpyHostToDevice));
		clock_gettime(CLOCK_REALTIME, &time1);
		timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

		// Run kernel
		clock_gettime(CLOCK_REALTIME, &time0);
		cu_BVHgpu_trace<<<grid, threads>>>(gpu_nNodes, cu_rays, nRaysF[rayBuffer], cu_gpu_primitives);
		clock_gettime(CLOCK_REALTIME, &time1);
		timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

		// Generate next rays while the kernel is running
		k2 = k + nRays;

		if (k2 < nRaysTotal)
		{
			clock_gettime(CLOCK_REALTIME, &time0);

			if ((nRaysTotal - k2) < nRays)
				nRaysF[!rayBuffer] = nRaysTotal - k2;

			for (unsigned int i = 0; i < nRaysF[!rayBuffer]; i++)
			{
				// Calculate buffer coordinates
				x = ((k2 + i) / kernelSize) % width;
				y = ((k2 + i) / kernelSize) / width;

				// Calculate kernel coordinates
				kernelIdx[0] = ((k2 + i) % kernelSize) % kernelDim[0];
				kernelIdx[1] = ((k2 + i) % kernelSize) / kernelDim[0];

				// Get camera ray
				camera.getRay(
					((float)x * 2.0f - (float)width) / (float)width,
					((float)y * 2.0f - (float)height) / (float)width,
					kernelDim, kernelIdx,
					rays[!rayBuffer][i].pos, rays[!rayBuffer][i].dir);

				// Reset distance
				rays[!rayBuffer][i].distance = -1.0f;
			}
			clock_gettime(CLOCK_REALTIME, &time1);
			timeGenRays += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
		}

		// Synchronize to wait for the kernel to finish
		clock_gettime(CLOCK_REALTIME, &time0);
		hipDeviceSynchronize();
		clock_gettime(CLOCK_REALTIME, &time1);
		timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

		// Copy rays from GPU
		clock_gettime(CLOCK_REALTIME, &time0);
		cutilSafeCall(hipMemcpy(rays[rayBuffer], cu_rays, sizeof(BVHgpu_ray) * nRaysF[rayBuffer], hipMemcpyDeviceToHost));
		clock_gettime(CLOCK_REALTIME, &time1);
		timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

		// Fill buffer
		clock_gettime(CLOCK_REALTIME, &time0);
		for (unsigned int i = 0; i < nRaysF[rayBuffer]; i++)
		{
			if (rays[rayBuffer][i].distance != -1.0f)
			{
				// Calculate buffer coordinates
				pos = (k + i) / kernelSize;

				// Accumulate buffer
				static float aux;
				//aux = 0.5f + 0.5f * sinf(rays[rayBuffer][i].distance * 10.0f);
				aux = rays[rayBuffer][i].distance * 0.03f;
				renderBuffer[pos].rgb[0] += aux;
				renderBuffer[pos].rgb[1] += aux;
				renderBuffer[pos].rgb[2] += aux;
				renderBuffer[pos].alpha += 1.0f;
			}
		}
		clock_gettime(CLOCK_REALTIME, &time1);
		timeFillBuffer += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
	}

	// Rescale buffer
	clock_gettime(CLOCK_REALTIME, &time0);
	for (unsigned int i = 0; i < width * height; i++)
	{
		renderBuffer[i].rgb[0] /= kernelSize;
		renderBuffer[i].rgb[1] /= kernelSize;
		renderBuffer[i].rgb[2] /= kernelSize;
		renderBuffer[i].alpha /= kernelSize;
	}
	clock_gettime(CLOCK_REALTIME, &time1);
	timeInit += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Calculate and show times */

    timeMemory = timeAllocation + timeHostToDevice + timeBindTextures + timeRaysToDevice + timeRaysToHost;
    timeCPU = timeInit + timeGenRays + timeFillBuffer + timeOthers;
    timeTotal = timeKernel + timeCPU + timeMemory;

    std::cout << "Time total             : " << timeTotal << std::endl;
    std::cout << "  Time CPU             : " << timeCPU << "\t(" << (100.0 * timeCPU / timeTotal) << "%)" << std::endl;
    std::cout << "    Time init          : " << timeInit << "\t(" << (100.0 * timeInit / timeTotal) << "%)" << std::endl;
    std::cout << "    Time gen rays      : " << timeGenRays << "\t(" << (100.0 * timeGenRays / timeTotal) << "%)" << std::endl;
    std::cout << "    Time fill buffer   : " << timeFillBuffer << "\t(" << (100.0 * timeFillBuffer / timeTotal) << "%)" << std::endl;
    std::cout << "    Time others        : " << timeOthers << "\t(" << (100.0 * timeOthers / timeTotal) << "%)" << std::endl;
    std::cout << "  Time memory          : " << timeMemory << "\t(" << (100.0 * timeMemory / timeTotal) << "%)" << std::endl;
    std::cout << "    Time allocation    : " << timeAllocation << "\t(" << (100.0 * timeAllocation / timeTotal) << "%)" << std::endl;
    std::cout << "    Time host to device: " << timeHostToDevice << "\t(" << (100.0 * timeHostToDevice / timeTotal) << "%)" << std::endl;
    std::cout << "    Time bind textures : " << timeBindTextures  << "\t(" << (100.0 * timeBindTextures / timeTotal) << "%)" << std::endl;
    std::cout << "    Time rays to device: " << timeRaysToDevice << "\t(" << (100.0 * timeRaysToDevice / timeTotal) << "%)" << std::endl;
    std::cout << "    Time rays to host  : " << timeRaysToHost << "\t(" << (100.0 * timeRaysToHost / timeTotal) << "%)" << std::endl;
    std::cout << "  Time Kernel          : " << timeKernel << "\t(" << (100.0 * timeKernel / timeTotal) << "%)" << std::endl;
    std::cout << "    Time parallel      : " << (timeKernel + timeGenRays) << "\t(" << (100.0 * (timeKernel + timeGenRays) / timeTotal) << "%)" << std::endl;


    	/* Free memory */

    cutilSafeCall(hipFree(cu_gpu_nodesMin));
    cutilSafeCall(hipFree(cu_gpu_nodesMax));
    cutilSafeCall(hipFree(cu_gpu_nodesSkipLocation));
	cutilSafeCall(hipFree(cu_gpu_primitives));
	cutilSafeCall(hipFree(cu_rays));
	delete [] rays[0];
	delete [] rays[1];
}
