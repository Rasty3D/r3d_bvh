#include "hip/hip_runtime.h"
/*
 * bvhgpu.cu
 *
 *  Created on: 07 Feb 2011
 *      Author: showroom
 */


/*
 * INCLUDES
 */

	/* CUDA */
#include <hip/hip_runtime.h>

	/* BVH */
#include "bvhcudainline.h"
#include "bvhgpu.h"


/*
 * TEXTURES
 */

texture<float4, 1, hipReadModeElementType> nodeTex_min;
texture<float4, 1, hipReadModeElementType> nodeTex_max;
texture<uint2, 1, hipReadModeElementType> nodeTex_skipLocation;


/*
 * NAMESPACES
 */

using namespace std;


/*
 * FUNCTIONS
 */

__device__ int cu_BVHgpu_primitiveIntersection(BVHgpu_ray *ray, unsigned char *primitive)
{
	// Variables
	float t;
	float pi[3];
	BVHPrimitive_triangle *triangle;
	BVHgpu_primitiveHeader *header = (BVHgpu_primitiveHeader*)primitive;
	unsigned char *data = primitive + sizeof(BVHgpu_primitiveHeader);

	if (header->type == BVHPRIMITIVE_TYPE_TRIANGLE)
	{
		// Get triangle data
		triangle = (BVHPrimitive_triangle*)data;

		// Calculate t
		t = ((triangle->v1[0] - ray->pos[0]) * triangle->normal[0] +
			 (triangle->v1[1] - ray->pos[1]) * triangle->normal[1] +
			 (triangle->v1[2] - ray->pos[2]) * triangle->normal[2]) /
			(ray->dir[0] * triangle->normal[0] +
			 ray->dir[1] * triangle->normal[1] +
			 ray->dir[2] * triangle->normal[2]);

		// Check direction
		if (t < 0.001f)
			return 0;

		if (ray->distance != -1.0f && t > ray->distance)
			return 0;

		// Check intersection
		pi[0] = ray->pos[0] + ray->dir[0] * t;
		pi[1] = ray->pos[1] + ray->dir[1] * t;
		pi[2] = ray->pos[2] + ray->dir[2] * t;

		if (((pi[0] - triangle->v1[0]) * triangle->n12[0] +
			 (pi[1] - triangle->v1[1]) * triangle->n12[1] +
			 (pi[2] - triangle->v1[2]) * triangle->n12[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v2[0]) * triangle->n23[0] +
			 (pi[1] - triangle->v2[1]) * triangle->n23[1] +
			 (pi[2] - triangle->v2[2]) * triangle->n23[2]) < 0.0f)
			return 0;

		if (((pi[0] - triangle->v3[0]) * triangle->n31[0] +
			 (pi[1] - triangle->v3[1]) * triangle->n31[1] +
			 (pi[2] - triangle->v3[2]) * triangle->n31[2]) < 0.0f)
			return 0;

		ray->distance = t;
		return 1;
	}

	return 0;
}

__device__ int cu_BVHgpu_aabbIntersection(int nodeIndex, BVHgpu_ray *ray, float *dirInv)
{
	// Calculate intersection
	float tpos[3];
	float tneg[3];
	float tmax[3];
	float tmin[3];
	float lmin;
	float lmax;
	float4 aabbmin = tex1Dfetch(nodeTex_min, nodeIndex);
	float4 aabbmax = tex1Dfetch(nodeTex_max, nodeIndex);

	tpos[0] = (aabbmax.x - ray->pos[0]) * dirInv[0];
	tpos[1] = (aabbmax.y - ray->pos[1]) * dirInv[1];
	tpos[2] = (aabbmax.z - ray->pos[2]) * dirInv[2];
	tneg[0] = (aabbmin.x - ray->pos[0]) * dirInv[0];
	tneg[1] = (aabbmin.y - ray->pos[1]) * dirInv[1];
	tneg[2] = (aabbmin.z - ray->pos[2]) * dirInv[2];

	tmax[0] = max(tpos[0], tneg[0]);
	tmax[1] = max(tpos[1], tneg[1]);
	tmax[2] = max(tpos[2], tneg[2]);
	tmin[0] = min(tpos[0], tneg[0]);
	tmin[1] = min(tpos[1], tneg[1]);
	tmin[2] = min(tpos[2], tneg[2]);

	lmin = max(tmin[0], max(tmin[1], tmin[2]));
	lmax = min(tmax[0], min(tmax[1], tmax[2]));

	if (ray->distance == -1.0f)
		return lmax > lmin && lmax > 0.0f;
	else
		return lmax > lmin && lmax > 0.0f && lmin < ray->distance;
}

__global__ void cu_BVHgpu_trace(const unsigned int gpu_nNodes, BVHgpu_ray *rays, const int nRays, unsigned char *primitives)
{
	// Normalize ray
	float rayModule;
	float dirInv[3];
	unsigned int nodeIndex = 0;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	uint2 skipLocation;

	if (idx >= nRays)
		return;

	if (rays[idx].distance == -2.0f)
		return;

	rayModule = sqrtf(
		rays[idx].dir[0] * rays[idx].dir[0] +
		rays[idx].dir[1] * rays[idx].dir[1] +
		rays[idx].dir[2] * rays[idx].dir[2]);
	rays[idx].dir[0] /= rayModule;
	rays[idx].dir[1] /= rayModule;
	rays[idx].dir[2] /= rayModule;

	// Inverse ray
	dirInv[0] = 1.0f / rays[idx].dir[0];
	dirInv[1] = 1.0f / rays[idx].dir[1];
	dirInv[2] = 1.0f / rays[idx].dir[2];

	// Tree traversal
	while (nodeIndex < gpu_nNodes)
	{
		skipLocation = tex1Dfetch(nodeTex_skipLocation, nodeIndex);

		if (cu_BVHgpu_aabbIntersection(nodeIndex, &rays[idx], dirInv))
		{
			if (skipLocation.y != BVHGPU_NOPRIMITIVE)
			{
				if (cu_BVHgpu_primitiveIntersection(&rays[idx], &primitives[skipLocation.y]))
					rays[idx].closestNode = nodeIndex;
			}

			nodeIndex++;
		}
		else
		{
			nodeIndex = skipLocation.x;
		}
	}
}

void cu_BVHgpu_render(
	// Scene definition
	unsigned int gpu_nPrimitives, unsigned int *gpu_primitiveLocations,
	unsigned int gpu_primitiveSize, unsigned char *gpu_primitives,
	unsigned int gpu_nNodes, float *gpu_nodesMin, float *gpu_nodesMax, unsigned int *gpu_nodesSkipLocation,
	// Render buffer
	unsigned int width, unsigned int height, BVH_gpuRenderBuffer *renderBuffer, void *scene, int samplesPerPixel,
	int lightDepth, int eyeDepth, float blurRadius,
	// Kernel control
	int nThreadsGPU, int nThreadsCPU, BVHgpu_bucket *bucket,
	// Stats
	BVHgpu_stats *stats)
{
		/* Variables */

	BVHgpu_renderState state;

	int pos;

	dim3 threads(1, 1);
	dim3 grid(1, 1);

	unsigned char *cu_gpu_primitives;
	BVHgpu_ray *cu_rays;
	float *cu_gpu_nodesMin;
	float *cu_gpu_nodesMax;
	unsigned int *cu_gpu_nodesSkipLocation;

	struct timespec time0, time1;

	BVHgpu_threadData *threadDataCPU = new BVHgpu_threadData[nThreadsCPU];
	pthread_t *threadsCPU = new pthread_t[nThreadsCPU];


		/* Configure textures */

	// Min
	nodeTex_min.normalized = 0;
	nodeTex_min.filterMode = hipFilterModePoint;
	nodeTex_min.addressMode[0] = hipAddressModeClamp;

	// Max
	nodeTex_max.normalized = 0;
	nodeTex_max.filterMode = hipFilterModePoint;
	nodeTex_max.addressMode[0] = hipAddressModeClamp;

	// Skip index and primitive location
	nodeTex_skipLocation.normalized = 0;
	nodeTex_skipLocation.filterMode = hipFilterModePoint;
	nodeTex_skipLocation.addressMode[0] = hipAddressModeClamp;


		/* Init stats */

	stats->timeInit = 0.0;
	stats->timeAllocation = 0.0;
	stats->timeHostToDevice = 0.0;
	stats->timeRaysToDevice = 0.0;
	stats->timeRaysToHost = 0.0;
	stats->timeKernel = 0.0;
	stats->timeOthers = 0.0;
	stats->timeBindTextures = 0.0;
	stats->timeGenRays = 0.0;
	stats->timeFillBuffer = 0.0;


		/* Init things */

	clock_gettime(CLOCK_REALTIME, &time0);

	// Bucket
	state.bucket = *bucket;
	state.bucketSize = state.bucket.size[0] * state.bucket.size[1];

	// Paths
	state.eyeSegmentLength = eyeDepth;
	state.lightSegmentLength = lightDepth;
	state.nRaysMax = state.bucketSize * state.eyeSegmentLength;
	state.paths = new BVHBidi[state.bucketSize];

	// Rays
	state.rays[0] = new BVHgpu_ray[state.nRaysMax];
	state.rays[1] = new BVHgpu_ray[state.nRaysMax];
	state.nRays[0] = 0;
	state.nRays[1] = 0;

	// Primitives
	state.primitives = gpu_primitives;
	state.nodesSkipLocation = gpu_nodesSkipLocation;

	// Dimensions
	state.width = width;
	state.height = height;

	// Scene
	state.scene = scene;

	// Render buffer
	state.renderBuffer = renderBuffer;

	// Samples per pixel
	state.samplesPerPixel = samplesPerPixel;

	// Blur radius
	state.blurRadius = blurRadius;

	// Render buffer
	for (int x = state.bucket.pos[0]; x < state.bucket.pos[0] + state.bucket.size[0]; x++)
	{
		for (int y = state.bucket.pos[1]; y < state.bucket.pos[1] + state.bucket.size[1]; y++)
		{
			pos = x + y * width;

			// Reset buffer
			renderBuffer[pos].rgb[0] = 0.0f;
			renderBuffer[pos].rgb[1] = 0.0f;
			renderBuffer[pos].rgb[2] = 0.0f;
			renderBuffer[pos].alpha = 0.0f;
		}
	}

	clock_gettime(CLOCK_REALTIME, &time1);
	stats->timeInit += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Copy data to GPU memory */

	// Memory allocation
	clock_gettime(CLOCK_REALTIME, &time0);
	cuda_safeCall(hipMalloc((void**)&cu_gpu_primitives, gpu_primitiveSize));
	cuda_safeCall(hipMalloc((void**)&cu_gpu_nodesMin, gpu_nNodes * 4 * sizeof(float)));
	cuda_safeCall(hipMalloc((void**)&cu_gpu_nodesMax, gpu_nNodes * 4 * sizeof(float)));
	cuda_safeCall(hipMalloc((void**)&cu_gpu_nodesSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int)));
	cuda_safeCall(hipMalloc((void**)&cu_rays, sizeof(BVHgpu_ray) * state.nRaysMax));
	clock_gettime(CLOCK_REALTIME, &time1);
	stats->timeAllocation += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Copy data to graphic card
	clock_gettime(CLOCK_REALTIME, &time0);
	cuda_safeCall(hipMemcpy(cu_gpu_primitives, gpu_primitives, gpu_primitiveSize, hipMemcpyHostToDevice));
	cuda_safeCall(hipMemcpy(cu_gpu_nodesMin, gpu_nodesMin, gpu_nNodes * 4 * sizeof(float), hipMemcpyHostToDevice));
	cuda_safeCall(hipMemcpy(cu_gpu_nodesMax, gpu_nodesMax, gpu_nNodes * 4 * sizeof(float), hipMemcpyHostToDevice));
	cuda_safeCall(hipMemcpy(cu_gpu_nodesSkipLocation, gpu_nodesSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int), hipMemcpyHostToDevice));
	clock_gettime(CLOCK_REALTIME, &time1);
	stats->timeHostToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

	// Bind data to textures
	clock_gettime(CLOCK_REALTIME, &time0);
	hipChannelFormatDesc channelDescMin = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, nodeTex_min, cu_gpu_nodesMin, channelDescMin, gpu_nNodes * 4 * sizeof(float));
	hipChannelFormatDesc channelDescMax = hipCreateChannelDesc<float4>();
	hipBindTexture(NULL, nodeTex_max, cu_gpu_nodesMax, channelDescMax, gpu_nNodes * 4 * sizeof(float));
	hipChannelFormatDesc channelDescSkipLocation = hipCreateChannelDesc<uint2>();
	hipBindTexture(NULL, nodeTex_skipLocation, cu_gpu_nodesSkipLocation, channelDescSkipLocation, gpu_nNodes * 2 * sizeof(unsigned int));
	clock_gettime(CLOCK_REALTIME, &time1);
	stats->timeBindTextures += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);


		/* Launch threads */

	BVHgpu_pathtracing_launchThreads(&state, nThreadsCPU, threadDataCPU, threadsCPU);


		/* Render bucket */

	for (int s = 0; s < samplesPerPixel; s++)
	{
		// Init phase
		state.phase = PHASE_EYE_RAY;
		state.eyeDepth = 0;
		state.lightDepth = 0;
		state.subpathId = 0;
		state.subpathMax = state.paths[0].getSubpathNumber();
		state.subpathStep = state.paths[0].getSubpathStep();
		state.rayBuffer = 0;

		// Reset paths
		for (pos = 0; pos < state.bucketSize; pos++)
			state.paths[pos].reset();

		while (1)
		{
			// No in the first step (there is no rays in the buffer)
			if (state.nRays[!state.rayBuffer] > 0)
			{
				// Configure the thread grid
				threads.x = nThreadsGPU;
				grid.x = iDivUp(state.nRays[!state.rayBuffer], nThreadsGPU);

				// Copy rays to GPU
				clock_gettime(CLOCK_REALTIME, &time0);
				cuda_safeCall(hipMemcpy(cu_rays, state.rays[!state.rayBuffer], sizeof(BVHgpu_ray) * state.nRays[!state.rayBuffer], hipMemcpyHostToDevice));
				clock_gettime(CLOCK_REALTIME, &time1);
				stats->timeRaysToDevice += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

				// Run kernel
				clock_gettime(CLOCK_REALTIME, &time0);
				cu_BVHgpu_trace<<<grid, threads>>>(gpu_nNodes, cu_rays, state.nRays[!state.rayBuffer], cu_gpu_primitives);
				clock_gettime(CLOCK_REALTIME, &time1);
				stats->timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
			}

			clock_gettime(CLOCK_REALTIME, &time0);
			if (!BVHgpu_pathtracing(&state, nThreadsCPU, threadDataCPU, threadsCPU))
				break;
			clock_gettime(CLOCK_REALTIME, &time1);
			stats->timeGenRays += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

			// No in the first step (there is no rays in the buffer)
			if (state.nRays[!state.rayBuffer] > 0)
			{
				// Synchronize to wait for the kernel to finish
				clock_gettime(CLOCK_REALTIME, &time0);
				hipDeviceSynchronize();
				clock_gettime(CLOCK_REALTIME, &time1);
				stats->timeKernel += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);

				// Copy rays from GPU
				clock_gettime(CLOCK_REALTIME, &time0);
				cuda_safeCall(hipMemcpy(state.rays[!state.rayBuffer], cu_rays, sizeof(BVHgpu_ray) * state.nRays[!state.rayBuffer], hipMemcpyDeviceToHost));
				clock_gettime(CLOCK_REALTIME, &time1);
				stats->timeRaysToHost += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
			}

			// Swap buffers
			state.rayBuffer = !state.rayBuffer;

			// Next phase
			clock_gettime(CLOCK_REALTIME, &time0);
			if (state.phase == PHASE_EYE_RAY)
			{
				state.phase = PHASE_LIGHT_RAY;
				state.eyeDepth++;

				if (state.eyeDepth >= state.eyeSegmentLength && state.lightDepth >= state.lightSegmentLength)
					state.phase = PHASE_SUBPATH;
			}
			else if (state.phase == PHASE_LIGHT_RAY)
			{
				state.phase = PHASE_EYE_RAY;
				state.lightDepth++;

				if (state.eyeDepth >= state.eyeSegmentLength && state.lightDepth >= state.lightSegmentLength)
					state.phase = PHASE_SUBPATH;
			}
			else if (state.phase == PHASE_SUBPATH)
			{
				state.subpathId += state.subpathStep;

				if (state.subpathId > state.subpathMax + state.subpathStep)
				{
					clock_gettime(CLOCK_REALTIME, &time1);
					stats->timeOthers += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
					break;
				}
			}
			else
			{
				break;
			}
			clock_gettime(CLOCK_REALTIME, &time1);
			stats->timeOthers += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
		}

		// Fill buffer
		clock_gettime(CLOCK_REALTIME, &time0);
		state.phase = PHASE_FILL_BUFFER;
		if (!BVHgpu_pathtracing(&state, nThreadsCPU, threadDataCPU, threadsCPU))
			break;
		clock_gettime(CLOCK_REALTIME, &time1);
		stats->timeFillBuffer += (double)(time1.tv_sec - time0.tv_sec) + 1E-9 * (double)(time1.tv_nsec - time0.tv_nsec);
	}


		/* Stop threads */

	BVHgpu_pathtracing_stopThreads(&state, nThreadsCPU, threadDataCPU, threadsCPU);


		/* Calculate stats */

    stats->timeMemory = stats->timeAllocation + stats->timeHostToDevice + stats->timeBindTextures + stats->timeRaysToDevice + stats->timeRaysToHost;
    stats->timeCPU = stats->timeInit + stats->timeGenRays + stats->timeFillBuffer + stats->timeOthers;
    stats->timeTotal = stats->timeKernel + stats->timeCPU + stats->timeMemory;


    	/* Free memory */

    cuda_safeCall(hipFree(cu_gpu_nodesMin));
    cuda_safeCall(hipFree(cu_gpu_nodesMax));
    cuda_safeCall(hipFree(cu_gpu_nodesSkipLocation));
	cuda_safeCall(hipFree(cu_gpu_primitives));
	cuda_safeCall(hipFree(cu_rays));
	delete [] state.paths;
	delete [] state.rays[0];
	delete [] state.rays[1];
	delete [] threadDataCPU;
	delete [] threadsCPU;
}
